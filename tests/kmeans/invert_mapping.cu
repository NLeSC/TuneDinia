#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void invert_mapping(float *input,			/* original */
							   float *output,			/* inverted */
							   int npoints,				/* npoints */
							   int nfeatures)			/* nfeatures */
{
	int point_id = threadIdx.x + block_size_x*blockIdx.x;	/* id of thread */
	int i;

	if(point_id < npoints){
		for(i=0;i<nfeatures;i++){
			output[point_id + npoints*i] = input[point_id*nfeatures + i];
    }
	}
	return;
}

#endif 