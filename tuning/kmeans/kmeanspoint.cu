#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#define GPU_DELTA_REDUCTION
#define GPU_NEW_CENTER_REDUCTION

__global__ void
kmeansPoint(float  *features,			/* in: [npoints*nfeatures] */
            int     nfeatures,
            int     npoints,
            int     nclusters,
            int    *membership,
			float  *clusters, int* block_deltas,
			float* block_clusters,
			float* feature_flipped_d) 
{

	// block ID
	const unsigned int block_id = gridDim.x*blockIdx.y+blockIdx.x;
	// point/thread ID  
	const unsigned int point_id = block_id*block_size_x*block_size_y + threadIdx.x;

	int  index = -1;

	if (point_id < npoints)
	{
		int i, j;
		float min_dist =  1E+37;
		float dist;													/* distance square between a point to cluster center */
		
		/* find the cluster center id with min distance to pt */
		for (i=0; i<nclusters; i++) {
			int cluster_base_index = i*nfeatures;					/* base index of cluster centers for inverted array */			
			float ans=0.0;												/* Euclidean distance sqaure */

			for (j=0; j < nfeatures; j++)
			{					
				int addr = point_id + j*npoints;					/* appropriate index of data point */
				float diff = (features[addr] -  //t_features[addr]
							  clusters[cluster_base_index + j]);	/* distance between a data point to cluster centers */
				ans += diff*diff;									/* sum of squares */
			}
			dist = ans;		

			/* see if distance is smaller than previous ones:
			if so, change minimum distance and save index of cluster center */
			if (dist < min_dist) {
				min_dist = dist;
				index = i;
			}
		}
	}
	#ifdef GPU_DELTA_REDUCTION
    // count how many points are now closer to a different cluster center	
	__shared__ int deltas[block_size_x*block_size_y];
	if(threadIdx.x < block_size_x*block_size_y) {
		deltas[threadIdx.x] = 0;
	}
#endif
	if (point_id < npoints)
	{
#ifdef GPU_DELTA_REDUCTION
		/* if membership changes, increase delta by 1 */
		if (membership[point_id] != index) {
			deltas[threadIdx.x] = 1;
		}
#endif
		/* assign the membership to object point_id */
		membership[point_id] = index;
	}

#ifdef GPU_DELTA_REDUCTION
	// make sure all the deltas have finished writing to shared memory
	__syncthreads();

	// now let's count them
	// primitve reduction follows
	unsigned int threadids_participating = (block_size_x*block_size_y) / 2;
	for(;threadids_participating > 1; threadids_participating /= 2) {
   		if(threadIdx.x < threadids_participating) {
			deltas[threadIdx.x] += deltas[threadIdx.x + threadids_participating];
		}
   		__syncthreads();
	}
	if(threadIdx.x < 1)	{deltas[threadIdx.x] += deltas[threadIdx.x + 1];}
	__syncthreads();
		// propagate number of changes to global counter
	if(threadIdx.x == 0) {
		block_deltas[blockIdx.y * gridDim.x + blockIdx.x] = deltas[0];
		//printf("original id: %d, modified: %d\n", blockIdx.y*gridDim.x+blockIdx.x, blockIdx.x);
		
	}

#endif


#ifdef GPU_NEW_CENTER_REDUCTION
	int center_id = threadIdx.x / nfeatures;    
	int dim_id = threadIdx.x - nfeatures*center_id;

	__shared__ int new_center_ids[block_size_x*block_size_y];

	new_center_ids[threadIdx.x] = index;
	__syncthreads();

	/***
	determine which dimension calculte the sum for
	mapping of threads is
	center0[dim0,dim1,dim2,...]center1[dim0,dim1,dim2,...]...
	***/ 	

	int new_base_index = (point_id - threadIdx.x)*nfeatures + dim_id;
	float accumulator = 0.f;

	if(threadIdx.x < nfeatures * nclusters) {
		// accumulate over all the elements of this threadblock 
		for(int i = 0; i< (block_size_x*block_size_y); i++) {
			float val = feature_flipped_d[new_base_index+i*nfeatures];
			if(new_center_ids[i] == center_id) 
				accumulator += val;
		}
	
		// now store the sum for this threadblock
		/***
		mapping to global array is
		block0[center0[dim0,dim1,dim2,...]center1[dim0,dim1,dim2,...]...]block1[...]...
		***/
		block_clusters[(blockIdx.y*gridDim.x + blockIdx.x) * nclusters * nfeatures + threadIdx.x] = accumulator;
	}
#endif
}
#endif // #ifndef _KMEANS_CUDA_KERNEL_H_