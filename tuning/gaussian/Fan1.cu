#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <stdio.h>
#include <assert.h>

__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t)
{   
	if(threadIdx.x + blockIdx.x * block_size_x >= Size-1-t) return;
	*(m_cuda+Size*(block_size_x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(block_size_x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}
#endif